
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>

void input(char *input_filename);
void output(char *output_filename);

__device__ float _max(float a, float b) { return a > b ? a : b; }
__device__ float _min(float a, float b) { return a < b ? a : b; }

int B, N, d;
float *Q, *K, *V, *O;
#define br 32
#define bc 32
#define d_offset 65

void input(char *input_filename) {
    FILE *file = fopen(input_filename, "rb");

    fread(&B, sizeof(int), 1, file);
    fread(&N, sizeof(int), 1, file);
    fread(&d, sizeof(int), 1, file);

    Q = (float *)malloc(B * N * d * sizeof(float));
    K = (float *)malloc(B * N * d * sizeof(float));
    V = (float *)malloc(B * N * d * sizeof(float));
    O = (float *)malloc(B * N * d * sizeof(float));

    for (int i = 0; i < B; i++) {
        fread(Q + (i * N * d), sizeof(float), N * d, file);
        fread(K + (i * N * d), sizeof(float), N * d, file);
        fread(V + (i * N * d), sizeof(float), N * d, file);
    }
    memset(O, 0x00, B * N * d * sizeof(float));

    fclose(file);
}

void output(char *output_filename) {
    FILE *file = fopen(output_filename, "wb");

    fwrite(O, sizeof(float), B * N * d, file);

    free(Q);
    free(K);
    free(V);
    free(O);

    fclose(file);
}

__global__ void flash_attention(float *q, float *k, float *v, float *o, float* l, float* m, int d, int tc, int N){
    for (int j = 0; j < tc; j++){
        // Load k and v to shared memory
        __shared__ float kj[bc*d_offset];
        __shared__ float vj[bc*d_offset];
        int round = d / bc;
        for(int i=0;i<round;i++){
            kj[threadIdx.y*d_offset+threadIdx.x+i*bc] = k[j*bc*d+threadIdx.y*d+threadIdx.x+i*bc];
            vj[threadIdx.y*d_offset+threadIdx.x+i*bc] = v[j*bc*d+threadIdx.y*d+threadIdx.x+i*bc];
        }

        // Shared memory allocations
        __shared__ float sij[br*bc];   // S = QK^T * scalar
        __shared__ float pij[br*bc];   // P = softmax(S)
        __shared__ float qi[br*d_offset];
        __shared__ float oi[br*d_offset];
        __shared__ float li[br];
        __shared__ float mi[br];
        __shared__ float mij[br];
        __shared__ float lij[br];

        // Load Q and O from global memory
        for(int round=0; round<d/bc; round++){
            qi[threadIdx.y*d_offset+threadIdx.x+round*bc] = q[blockIdx.x*br*d+threadIdx.y*d+threadIdx.x+round*bc];
            oi[threadIdx.y*d_offset+threadIdx.x+round*bc] = o[blockIdx.x*br*d+threadIdx.y*d+threadIdx.x+round*bc];
        }

        if(threadIdx.x == 0){
            li[threadIdx.y] = l[blockIdx.x*br+threadIdx.y];
            mi[threadIdx.y] = m[blockIdx.x*br+threadIdx.y];
        }
        __syncthreads();

        // Inline QKDotAndScalar
        {
            int row = threadIdx.y;
            int col = threadIdx.x;
            float val = 0.0f;
            for (int t = 0; t < d; t++) {
                val += qi[row * d_offset + t] * kj[col * d_offset + t];
            }
            sij[row * bc + col] = val * (1.0f / sqrtf((float)d));
        }
        __syncthreads();

        // Inline RowMax
        {
            int row = threadIdx.y;
            if (threadIdx.x == 0) {
                float max_val = sij[row * bc];
                for (int i = 0; i < bc; i++) {
                    max_val = _max(max_val, sij[row * bc + i]);
                }
                mij[row] = max_val;
            }
        }
        __syncthreads();

        // Inline MinusMaxAndExp
        {
            int row = threadIdx.y;
            int col = threadIdx.x;
            pij[row * bc + col] = expf(sij[row * bc + col] - mij[row]);
        }
        __syncthreads();

        // Inline RowSum
        {
            int row = threadIdx.y;
            if (threadIdx.x == 0) {
                float sum_val = 0.0f;
                for (int i = 0; i < bc; i++) {
                    sum_val += pij[row * bc + i];
                }
                lij[row] = sum_val;
            }
        }
        __syncthreads();

        // Inline UpdateMiLiOi
        {
            __shared__ float li_new[br];
            int i = threadIdx.y;
            int jx = threadIdx.x;

            // Compute mi_new, li_new
            if(jx == 0){ 
                li_new[i] = expf(mi[i] - 0) * li[i] + expf(mij[i] - 0) * lij[i];
            }
            // __syncthreads();

            // Update Oi
            for(int r =0; r < d/bc; r++){
                float pv = 0.0F;
                for (int t = 0; t < bc; t++) {
                    pv += pij[i * bc + t] * vj[t * d_offset + jx+r*bc];
                }
                // Weighted combination for oi
                oi[i * d_offset + jx+r*bc] = (li[i] * expf(mi[i] - 0) * oi[i * d_offset + jx+r*bc] 
                                             + expf(mij[i] - 0) * pv) / li_new[i];
            }
            // __syncthreads();

            // Update mi, li
            if(jx == 0){
                mi[i] = 0;
                li[i] = li_new[i];
            }
        }
        // __syncthreads();

        // Write back O, l, m
        for(int round=0; round<d/bc; round++){
            o[blockIdx.x*br*d+threadIdx.y*d+threadIdx.x+round*bc] = oi[threadIdx.y*d_offset+threadIdx.x+round*bc];
        }
        if(threadIdx.x == 0){
            l[blockIdx.x*br+threadIdx.y] = li[threadIdx.y];
            m[blockIdx.x*br+threadIdx.y] = mi[threadIdx.y];
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <input_filename> <output_filename>\n", argv[0]);
        return 1;
    }

    input(argv[1]);

    float *device_Q;
    float *device_K;
    float *device_V;
    float *device_O;
    float *device_l;
    float *device_m;

    //copy data to device
    hipHostRegister(Q, sizeof(float)*N*B*d, hipHostRegisterDefault);
    hipHostRegister(K, sizeof(float)*N*B*d, hipHostRegisterDefault);
    hipHostRegister(V, sizeof(float)*N*B*d, hipHostRegisterDefault);
    hipMalloc(&device_Q, sizeof(float)*N*B*d);
    hipMalloc(&device_K, sizeof(float)*N*B*d);
    hipMalloc(&device_V, sizeof(float)*N*B*d);
    hipMalloc(&device_O, sizeof(float)*N*B*d);
    hipMalloc(&device_l, sizeof(float)*N*B);
    hipMalloc(&device_m, sizeof(float)*N*B);
    hipMemcpy(device_Q, Q, sizeof(float)*N*B*d, hipMemcpyHostToDevice);
    hipMemcpy(device_K, K, sizeof(float)*N*B*d, hipMemcpyHostToDevice);
    hipMemcpy(device_V, V, sizeof(float)*N*B*d, hipMemcpyHostToDevice);

    hipMemset(device_O, 0, sizeof(float)*N*B*d);
    hipMemset(device_l, 0, sizeof(float)*N*B);
    hipMemset(device_m, FLT_MIN, sizeof(float)*N*B);

    int tr = N / br, tc = N / bc;
    dim3 blockPerGrid(tr);
    dim3 threadPerBlock(br, bc);

    for (int i = 0; i < B; i++) {
        flash_attention<<<blockPerGrid, threadPerBlock>>>(
            device_Q + (i * N * d), 
            device_K + (i * N * d), 
            device_V + (i * N * d), 
            device_O + (i * N * d),
            device_l + (i * N),
            device_m + (i * N),
            d, tc, N
        );
    }

    //copy data back to host
    hipMemcpy(O, device_O, sizeof(float)*N*B*d, hipMemcpyDeviceToHost);
    
    output(argv[2]);

    return 0;
}
